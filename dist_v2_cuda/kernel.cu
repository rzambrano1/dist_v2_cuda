#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <math.h>
#include <stdio.h>
#include "kernel.h"
#define TPB 32

__device__
float distance(float x1, float x2) {
	return sqrt((x2 - x1)*(x2 - x1));
}

__global__
void distanceKernel(float *d_out, float *d_in, float ref) {
	
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	const float x = d_in[i];

	d_out[i] = distance(x, ref);

	printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}

void distanceArray(float* out, float* in, float ref, int len) {

	// Declare pointers to device arrays
	float* d_in = 0; // Pointers to null, addresses no object
	float* d_out = 0;

	// Allocate memory for device arrays
	hipMalloc(&d_in, len * sizeof(float)); // The parameters of hipMalloc() are: devPtr [Pointer to allocated device memory] and size [Requested allocation size in bytes]
	hipMalloc(&d_out, len * sizeof(float));

	// Copy input data from host to device
	hipMemcpy(d_in, in, len * sizeof(float), hipMemcpyHostToDevice); // Parameters: dst [Destination memory address], src [Source memory address], 
	                                                                   //             count [Size in bytes to copy], kind [Type of transfer]

	// Launch kernel to compute and store distance values
	distanceKernel <<<dim3(len / TPB, 1, 1), dim3(TPB, 1, 1) >>> (d_out, d_in, ref);

	// Cpy results from device to host
	hipMemcpy(out, d_out, len * sizeof(float), hipMemcpyDeviceToHost);

	// Free memory allocated for device arrays
	hipFree(d_in);
	hipFree(d_out);
}
